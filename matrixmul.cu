
/* Multiplicación de matrices: P = M * N.
 * código Host.
 */

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

// includes, project
#include <cutil.h>

// includes, kernels
#include "matrixmul_kernel.cu"

#include "assist.h"

#define ERROR_CHECK { hipError_t err; \
  if ((err = hipGetLastError()) != hipSuccess) { \
    printf("CUDA error: %s, line %d\n", hipGetErrorString(err), __LINE__);}}

////////////////////////////////////////////////////////////////////////////////
// declaration, forward
void runTest(int argc, char** argv);

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int
main(int argc, char** argv)
{
    runTest(argc, argv);
}

////////////////////////////////////////////////////////////////////////////////
//! Run a simple test for CUDA
////////////////////////////////////////////////////////////////////////////////
void
runTest(int argc, char** argv)
{
    bool if_quiet = true;
    unsigned int timer_compute = 0;
    unsigned int timer_memory = 0;
    int i, j;
    char *matrix_id = NULL, *input_fn = NULL, *gold_fn = NULL;
    float * deviceM = NULL, * deviceN = NULL, * deviceP = NULL;
    int Mw = 0, Mh = 0, Nw = 0, Nh = 0, Pw = 0, Ph = 0;
    int block_size = 0;

    if (argc == 2) {
        matrix_id = strdup(argv[1]);

    } else {
        fprintf(stderr, "Error: Wrong input parameter numbers.\n");
        fprintf(stderr, "Usage:\n"
                        "$> ./lab2-matrixmul <8, 128, 512, 3072, 4096>\n"
                        "Examples:\n"
                        "      $> ./lab2-matrixmul 128\n"
                        );
        exit(1);
    }

    // Nota: Las tamaños de la matriz (width-ancho y height-alto) deben ser múltiplos del tamaño de bloque.
    if (!strcmp(matrix_id, "8")) {
        Mw = Mh = Nw = Nh = Pw = Ph = 8;
        block_size = 2; // numero de threads por bloque = block_size^2
        input_fn = strdup("matrix_8.bin");
        gold_fn = strdup("matrix_8.gold");
        if_quiet = false; // If not display matrix contents
    } else
    if (!strcmp(matrix_id, "128")) {
        Mw = Mh = Nw = Nh = Pw = Ph = 128;
        block_size = 16; // numero de threads por bloque = block_size^2
        input_fn = strdup("matrix_128.bin");
        gold_fn = strdup("matrix_128.gold");
        if_quiet = true; // If not display matrix contents
    } else
    if (!strcmp(matrix_id, "512")) {
        Mw = Mh = Nw = Nh = Pw = Ph = 512;
        block_size = 16; // numero de threads por bloque = block_size^2
        input_fn = strdup("matrix_512.bin");
        gold_fn = strdup("matrix_512.gold");
        if_quiet = true; // If not display matrix contents
    } else
    if (!strcmp(matrix_id, "3072")) {
        Mw = Mh = Nw = Nh = Pw = Ph = 3072;
        block_size = 16; // numero de threads por bloque = block_size^2
        input_fn = strdup("matrix_3072.bin");
        gold_fn = strdup("matrix_3072.gold");
        if_quiet = true; // If not display matrix contents
    } else
    if (!strcmp(matrix_id, "4096")) {
        Mw = Mh = Nw = Nh = Pw = Ph = 4096;
        block_size = 32; // numero de threads por bloque = block_size^2
        input_fn = strdup("matrix_4096.bin");
        gold_fn = strdup("matrix_4096.gold");
        if_quiet = true; // If not display matrix contents
    } else {
        printf("***Error en %s: %d: ID de la matriz no definido.\n",
            __FILE__, __LINE__);
        printf("   Deberías añadirlo al código fuente.\n");
        printf("   Valores actuales son 8, 128, 512, 3072, 4096.\n");
        exit(1);
    }

    printf("Nombre del fichero de entrada de matrices: %s\n", input_fn);

    // -----------------------------------------------------------------------
    // Setup host side
    // -----------------------------------------------------------------------

    int rc = GenMatrixFile(input_fn, Mw, Mh, if_quiet);
	
    printf("Preparacion de la parte host y lanzamiento del kernel:\n");

    // Reserva de memoria host para matrices M y N
    printf("  Reserva de memoria host para matrices M y N.\n");
    printf("    M: %d x %d\n", Mw, Mh);
    printf("    N: %d x %d\n", Nw, Nh);
    unsigned int size_M = Mw * Mh;
    unsigned int mem_size_M = sizeof(float) * size_M;
    float* hostM = (float*) malloc(mem_size_M);
    unsigned int size_N = Nw * (Nh);
    unsigned int mem_size_N = sizeof(float) * size_N;
    float* hostN = (float*) malloc(mem_size_N);

    // Reserva de memoria para resultados en el host
    printf("  Reserva de memoria para el resultado en la parte host.\n");
    unsigned int size_P = Pw * Ph;
    unsigned int mem_size_P = sizeof(float) * size_P;
    float* hostP = (float*) malloc(mem_size_P);

    // Initialize the input matrices.
    printf("  Inicializa las matrices de entrada.\n");
    unsigned int * matrix = ReadMatrixFile(input_fn, Pw, Ph, if_quiet);
    for (i = 0; i < Mw; i++)
        for (j = 0; j < Nw; j++)
	        hostM[i * Mw + j] = hostN[i * Mw + j] = (float) matrix[i*Mw + j];
    
    free(matrix); 
    matrix = NULL;

    // ===================================================================
    //  Parte 1 de la solución:
    //  Reservar device memory para las matrices de entrada.
    //  Copiar memoria desde el host a la device.
    // ===================================================================  

    CUT_SAFE_CALL(cutCreateTimer(&timer_memory));
    CUT_SAFE_CALL(cutStartTimer(timer_memory));

    printf("  Reservar memoria device.\n");
    CUDA_SAFE_CALL(hipMalloc((void **) &deviceM, mem_size_M ));
    CUDA_SAFE_CALL(hipMalloc((void **) &deviceN, mem_size_N ));

    printf("  Copiar host memory al device.\n");
    CUDA_SAFE_CALL(hipMemcpy(deviceM, hostM, mem_size_M, hipMemcpyHostToDevice ));
    CUDA_SAFE_CALL(hipMemcpy(deviceN, hostN, mem_size_N, hipMemcpyHostToDevice ));

    printf("  Reservar device memory para los resultados.\n");
    CUDA_SAFE_CALL(hipMalloc((void **) &deviceP, mem_size_P ));

    // Clean device memory
    hipMemset(deviceP, 0, mem_size_P);

    CUT_SAFE_CALL(cutStopTimer(timer_memory));

    // Fin de la solcuión parte 1
    // ===================================================================

    // ===================================================================
    // Comienzo de la solución parte 2
    // Inicializar los bloques de hilos y las diminesiones del grid
    // e invocar al CUDA kernel.
    // Puedes asumir que cada dimensión de la matriz es múltiplo de 
    // de tamaño del bloque definido.
    // ===================================================================

    printf("  Establecer los parametros de ejecución del kernel.\n");
    
    dim3 block(block_size,block_size);
    dim3 grid(Pw/block_size,Ph/block_size);

    printf("  # de hilos en un bloque: %d x %d (%d)\n",
        block.x, block.y, block.x * block.y);
    printf("  # de bloques en un grid: %d x %d (%d)\n",
        grid.x, grid.y, grid.x * grid.y);

    // ================================================
    // Inicializar las dimensiones de bloque y del grid aquí
    // ================================================

    printf("  Ejecutando el kernel...\n");

    // Comienza el timer_compute para calcular cuanto tiempo se consume en él.
    CUT_SAFE_CALL(cutCreateTimer(&timer_compute));
    CUT_SAFE_CALL(cutStartTimer(timer_compute));

    // Invocar el kernel CUDA aquí
    matrixMul<<<grid, block>>>(deviceP, deviceM, deviceN, Mh, Mw, Nw, block_size);

    // Asegurate que todos los hilos han terminado su trabajo antes de parar el timer 
    hipDeviceSynchronize();

    // Para el timer_compute
    CUT_SAFE_CALL(cutStopTimer(timer_compute));

    // Fin de la parte 2 de la solución
    // ===================================================================

    // Comprueba si la ejecución del kernel genera un error
    ERROR_CHECK
    CUT_CHECK_ERROR("Kernel execution failed");

    // ===================================================================
    // Comienza la parte 3 de la solución
    // Copiar los resultados devuelta al host
    // ===================================================================

    CUT_SAFE_CALL(cutStartTimer(timer_memory));

    printf("  Copiar los resultados de la device al host.\n");
    hipMemcpy(hostP, deviceP, mem_size_P, hipMemcpyDeviceToHost);

    CUT_SAFE_CALL(cutStopTimer(timer_memory));

    // Fin de la parte 3 de la solución
    // ===================================================================

    // ================================================
    // Mostrar la información de tiempo
    // ================================================

    printf("  GPU memory access time: %f (ms)\n",
        cutGetTimerValue(timer_memory));
    printf("  GPU computation time  : %f (ms)\n",
        cutGetTimerValue(timer_compute));
    printf("  GPU processing time   : %f (ms)\n",
        cutGetTimerValue(timer_compute) + cutGetTimerValue(timer_memory));
    CUT_SAFE_CALL(cutDeleteTimer(timer_memory));
    CUT_SAFE_CALL(cutDeleteTimer(timer_compute));

    // ================================================
    // Hacer la comparación
    // ================================================

    // Comprobamos los resultados si el tamaño de la matriz es <= 512x512
    //if (0) {
    printf("\nComprueba los resultados con los obtenidos por la CPU.\n");
    printf ("  Ejecutando la solución de referencia.\n");
    CUT_SAFE_CALL(cutCreateTimer(&timer_compute));
    CUT_SAFE_CALL(cutStartTimer(timer_compute));

    float* reference = (float*) malloc(mem_size_P);
    computeGold(reference, hostM, hostN, Mh, Mw, Nw);
    CUT_SAFE_CALL(cutStopTimer(timer_compute));

    printf("  Tiempo de procesamiento en la CPU   : %f (ms)\n",
        cutGetTimerValue(timer_compute));
    CUT_SAFE_CALL(cutDeleteTimer(timer_compute));

    printf("  CPU checksum: %g\n", CheckSum(reference, Mw, Nw));

    matrix = (unsigned int *) malloc (Pw * Ph * sizeof(unsigned int));
    for (i = 0; i < Ph; i++)
         for (j = 0; j < Pw; j++)
           matrix[i*Pw + j] = (unsigned int) reference[i*Pw + j];

    WriteMatrixFile("lab2-matrixmul.gold", matrix, Pw, Ph, 1);
    free (matrix); matrix = NULL;
    free(reference);

    printf("  GPU checksum: %g\n", CheckSum(hostP, Mw, Nw));

    /* Escribe la matriz C a un fichero binario de salida */
    matrix = (unsigned int *) malloc (Pw * Ph * sizeof(unsigned int));
    for (i = 0; i < Ph; i++)
        for (j = 0; j < Pw; j++)
	        matrix[i*Pw + j] = (unsigned int) hostP[i*Pw + j];
    WriteMatrixFile("lab2-matrixmul.bin", matrix, Pw, Ph, 1);
    free (matrix); matrix = NULL;

    if (Mw >= 3072 && Mh >= 3072) {
        CompareMatrixFile("lab2-matrixmul.bin", gold_fn, Pw, Ph, if_quiet);
    } else {
        CompareMatrixFile("lab2-matrixmul.bin", "lab2-matrixmul.gold",
            Pw, Ph, if_quiet);
    }
    
    // Limpia la memoria
    free(hostM); free(hostN); free(hostP);
    free(input_fn); free(gold_fn);

    // ===================================================================
    // Comienzo de la parte 4 de la solución
    // Librera device memory
    // ===================================================================

    CUDA_SAFE_CALL(hipFree(deviceM));
    CUDA_SAFE_CALL(hipFree(deviceN));
    CUDA_SAFE_CALL(hipFree(deviceP));

    // Fin de la parte 4 de la solución 
    // ===================================================================
}

