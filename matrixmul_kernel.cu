#include "hip/hip_runtime.h"
/* Matrix multiplication: P = M * N.
 * código device.
 */

#ifndef _MATRIXMUL_KERNEL_H_
#define _MATRIXMUL_KERNEL_H_

#include "matrixmul.h"

////////////////////////////////////////////////////////////////////////////////
//! Simple test kernel for device functionality
//! @param g_idata  input data in global memory
//! @param g_odata  output data in global memory
////////////////////////////////////////////////////////////////////////////////

    __global__ void
matrixMul(
    float* P, const float* M, const float* N,
    const int Mh, const int Mw, const int Nw,
    const int block_size)
{
    const int bx = blockIdx.x;
    const int by = blockIdx.y;

    const int tx = threadIdx.x;
    const int ty = threadIdx.y;

    float Psub = 0;
    int i = 0, indexM = 0, indexN = 0, indexP = 0;

    // ===================================================================
    // Comienza la parte 5 de la solución 
    // Determinar el índice de salida de cada hilo.
    // Calcular el producto de una fila de M y una columna de N 
    // para cada hilo.
    // Escribir el valor calculado en el índice adecuado de la matriz P.
    // ===================================================================

    // Indice del primer elemento de M cargado por este hilo del bloque
    indexM = by * Mw * block_size + ty * Mw;
	 

    // Indice del primer elemento de N procesado por cada bloque
    indexN = bx * block_size + tx;

    // índice de destino de la matriz
    // Establece el indexP para referenciar el elemento de salida de este hilo
    indexP =indexM + indexN;

    // Para cada índice desde [0, Width of M)
    for (i = 0; i < Mw; i++) {
//	if (tx == 0 && bx == 0) cuPrintf ("Entra en iteracion %d\n", i);
        // Multiplicar sus elementos correspondientes de M y N, y acumular en 
        // un suma parcial Psub.
        Psub += M[indexM] * N[indexN];

        // Actualiza los índices en M y N para la siguiente iteración 
        indexM = indexM + 1;
        indexN = indexN + Nw;
    }
    P[indexP] = Psub;

    // Fin de la parte 5 de la solución ============================================
}

#endif // #ifndef _MATRIXMUL_KERNEL_H_


